#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <math.h>
#include "layers.h"
#include "flappie_stdlib.h"
#include "util.h"
#include <hipblas.h>


#include <cblas.h>

#define GEMV

#    define _A 12102203.161561485f
#    define _B 1065353216.0f
#    define _BOUND 88.02969193111305
__device__ static inline float gpu_expf(float x) {
    x = fmaxf(-_BOUND, fminf(_BOUND, x));
    union {
        uint32_t i;
        float f;
    } value = {
    .i = (uint32_t) (_A * x + _B)};
    return value.f;
}

__device__ static inline float gpu_logisticf(float x) {
    return 1.0 / (1.0 + gpu_expf(-x));
}

__device__ static inline float gpu_tanhf(float x) {
    const float y = gpu_logisticf(x + x);
    return y + y - 1.0;
}


#define THREADS_PER_ROW 32
#define ROWS_PER_BLOCK  16
#define THREADS_PER_BLOCK  (ROWS_PER_BLOCK*THREADS_PER_ROW)
#define NUM_BLOCKS  (768/ROWS_PER_BLOCK)

__global__ void spmv_csr_vector_kernel_v1 ( const int num_rows , int num_cols, const float * data , const float * x , float * y) 
{
    __shared__ float vals [ROWS_PER_BLOCK * 32];

    const int thread_id   = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;    // global thread index
    const int thread_lane = threadIdx.x & (THREADS_PER_ROW - 1);          // thread index within the vector
    const int vector_id   = thread_id   /  THREADS_PER_ROW;               // global vector index
    const int vector_lane = threadIdx.x /  THREADS_PER_ROW;               // vector index within the block
    const int num_vectors = ROWS_PER_BLOCK * gridDim.x;                   // total number of active vectors
    int pos = threadIdx.x;
    //int pos = vector_id * 32 + thread_lane;

    for(int row = vector_id; row < num_rows; row += num_vectors)
    {
    		float sum = 0;
                for ( int jj = 0 + thread_lane ; jj < num_cols ; jj += THREADS_PER_ROW-1)
                  sum += data [ (row*num_cols)+jj ] * x [jj];
                // parallel reduction in shared memory
  		vals[pos] = sum;
        	if (thread_lane < 16) vals[pos] += vals[pos + 16];
        	if (thread_lane <  8) vals[pos] += vals[pos +  8];
        	if (thread_lane <  4) vals[pos] += vals[pos +  4];
        	if (thread_lane <  2) vals[pos] += vals[pos +  2];
        	if (thread_lane <  1) vals[pos] += vals[pos +  1];
                // first thread OF EACH WARP ACCUMULATES the result
                if ( thread_lane == 0)
                  y[row] += vals [ pos];
        }
}

__global__ void spmv_csr_vector_kernel_v2 ( const int num_rows , int num_cols, const float * data , const float * x , float * y) 
{
    __shared__ float vals [ROWS_PER_BLOCK * 32];

    const int thread_id   = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;    // global thread index
    const int thread_lane = threadIdx.x & (THREADS_PER_ROW - 1);          // thread index within the vector
    const int vector_id   = thread_id   /  THREADS_PER_ROW;               // global vector index
    const int vector_lane = threadIdx.x /  THREADS_PER_ROW;               // vector index within the block
    const int num_vectors = ROWS_PER_BLOCK * gridDim.x;                   // total number of active vectors
    int pos = threadIdx.x;

    for(int row = vector_id; row < num_rows; row += num_vectors)
    {
    		float sum = 0;
                for ( int jj = 0 + thread_lane ; jj < num_cols ; jj += THREADS_PER_ROW)
                  sum += data [ (row*num_cols)+jj ] * x [jj];
                // parallel reduction in shared memory
  		vals[pos] = sum;
        	if (THREADS_PER_ROW > 16) vals[threadIdx.x] = sum = sum + vals[threadIdx.x + 16];
        	if (THREADS_PER_ROW >  8) vals[threadIdx.x] = sum = sum + vals[threadIdx.x +  8];
        	if (THREADS_PER_ROW >  4) vals[threadIdx.x] = sum = sum + vals[threadIdx.x +  4];
        	if (THREADS_PER_ROW >  2) vals[threadIdx.x] = sum = sum + vals[threadIdx.x +  2];
        	if (THREADS_PER_ROW >  1) vals[threadIdx.x] = sum = sum + vals[threadIdx.x +  1];
                // first thread OF EACH WARP ACCUMULATES the result
                if ( thread_lane == 0)
                  y[row] += vals [ pos];
        }
}

__global__ void
 spmv_csr_scalar_kernel ( const int num_rows , const int cols , const float * data , const float * x , float * y)
 {
     int row = blockDim.x * blockIdx.x + threadIdx.x ;
     float dot = 0;
     if( row < num_rows )
     {
         for (int jj = 0 ; jj < cols ; jj ++)
             dot += data [ (row*cols)+jj ] * x[ jj ];
         y[ row ] += dot ;
     }
 }

flappie_matrix aes_grumod_linear_gpu( const_flappie_matrix X, const_flappie_matrix sW, flappie_matrix ostate, int backward, const_flappie_matrix W, const_flappie_matrix b, int layer) {
    RETURN_NULL_IF(NULL == X, NULL);
    assert(NULL != sW);

#ifdef GEMV
    hipError_t cudaStat ; // hipMalloc status
    hipblasStatus_t stat ; // CUBLAS functions status
#endif

    const size_t size = sW->nr;
    const size_t N = X->nc;
    assert(X->nr == 3 * size);
    assert(sW->nc == 3 * size);

    ostate = remake_flappie_matrix(ostate, size, N);
    flappie_matrix xColTmp = make_flappie_matrix(3 * size, 1);

    _Mat xCol, sCol1, sCol2, XnextBuf;
    memset(ostate->data.v, 0, ostate->nrq * sizeof(__m128));
    xCol = *X;
    sCol1 = *ostate;
    sCol2 = *ostate;
    xCol.nc = sCol1.nc = sCol2.nc = 1;
    if(backward) {
      xCol.data.v = X->data.v + (X->nc - 1) * X->nrq;
      sCol1.data.v = ostate->data.v;
      sCol2.data.v = ostate->data.v + (ostate->nc - 1) * ostate->nrq;
      grumod_step(&xCol, &sCol1, sW, xColTmp, &sCol2);
    }
    else {
      sCol1.data.v = ostate->data.v + ostate->nrq;
      sCol2.data.v = ostate->data.v;
      grumod_step(&xCol, &sCol1, sW, xColTmp, &sCol2);
    }

    flappie_matrix Xnext = remake_flappie_matrix(NULL, W->nc, ostate->nc);
    RETURN_NULL_IF(NULL == Xnext, NULL);
    for (size_t c = 0; c < Xnext->nc; c++) {
        memcpy(Xnext->data.v + c * Xnext->nrq, b->data.v, Xnext->nrq * sizeof(__m128));
    }

    float Cin[768], Cout[768], A[256*768];
    float *ostate_ptr;
    float *istate_ptr;

    memcpy(A, sW->data.f, 256*768*sizeof(float));

#ifdef GEMV
    float *d_a, *d_x, *d_y, *d_cin ;
    cudaStat = hipMalloc (( void **)& d_a , 768*256*sizeof(float)); // device // memory alloc for a
    cudaStat = hipMalloc (( void **)& d_x , 256*sizeof(float)); // device // memory alloc for x
    cudaStat = hipMalloc (( void **)& d_y , 768*sizeof(float)); // device // memory alloc for y
    cudaStat = hipMalloc (( void **)& d_cin , 768*sizeof(float)); // device // memory alloc for cin 
    hipMemcpy(d_a, A, 768*256*sizeof(float), hipMemcpyHostToDevice);
    float al =1.0f;
    float bet =1.0f;
#endif

    for (int i = 1; i < N; i++) {
        size_t index;
        // LOAD
        {
                if(backward) {
                        index = N - i - 1;
                        xCol.data.f = X->data.f + index * X->nr;
                        ostate_ptr = ostate->data.f + index * ostate->nr;
                        istate_ptr = ostate_ptr + 256;
        	}
                else {
                        index = i;
                        xCol.data.f = X->data.f + index * X->nr;
                        ostate_ptr = ostate->data.f + index * ostate->nr;
                        istate_ptr = ostate_ptr - 256;
                }

                memcpy(Cin, xCol.data.f, 768*sizeof(float));
                memcpy(Cout, xColTmp->data.f, 768*sizeof(float));
        }

        // COMPUTE
        {
                const size_t size = 256;
    		int M=768, N=256;
                memcpy(Cout, Cin, 768 * sizeof(float) );
                memset(Cout + size + size, 0, size *sizeof(float));

#ifdef GEMV
                hipMemcpy(d_x, istate_ptr, N*sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(d_y, Cout, M*sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(d_cin, Cin, M*sizeof(float), hipMemcpyHostToDevice);
                spmv_csr_vector_kernel_v1<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(M, N, d_a, d_x, d_y);
                //spmv_csr_vector_kernel_v2<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(M, N, d_a, d_x, d_y);
                //spmv_csr_scalar_kernel<<<1, 768>>>(M, N, d_a, d_x, d_y);

                hipMemcpy(Cout, d_y, M*sizeof(float), hipMemcpyDeviceToHost);
#else
                cblas_sgemv(CblasRowMajor, CblasNoTrans, 768, 256, 1.0, A, 256, istate_ptr, 1, 1.0, Cout, 1);
#endif

                for (size_t i = 0; i < size; i++) {
                        Cout[i] = LOGISTICF(Cout[i]);
                        Cout[size+i] = LOGISTICF(Cout[size+i]);
                        Cout[i+size+size] = TANHF(Cout[i+size] * Cout[i+size+size] + Cin[i+size+size]);
                        ostate_ptr[i] = (-1) * Cout[i] * Cout[i+size+size] + Cout[i+size+size];
                        ostate_ptr[i] = Cout[i] * istate_ptr[i] + ostate_ptr[i];
                        //ostate_ptr[i] = Cout[i];
                }

        }
        {
        }
    } // end of N iterations
    xColTmp = free_flappie_matrix(xColTmp);
    assert(validate_flappie_matrix (ostate, -1.0, 1.0, 0.0, true, __FILE__, __LINE__));

#ifdef GEMV
    hipFree (d_a );
    hipFree (d_x );
    hipFree (d_y );
#endif

    cblas_sgemm(CblasColMajor, CblasTrans, CblasNoTrans, W->nc, X->nc, W->nr, 1.0, W->data.f, W->stride, ostate->data.f, ostate->stride, 1.0, Xnext->data.f, Xnext->stride);
    return Xnext;
}

